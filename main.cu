#include <stdio.h>
#include <hip/hip_runtime.h>

#define N (1 << 20)
#define THREADS_PER_BLOCK 32

// Kernel definition
// Useful variables:
// blockDim.x: number of threads in a block
// blockIdx.x: index of the current block
// threadIdx.x: index of the current thread
__global__ void VecAdd(float *A, float *B,
                       float *C)
{
    // YOUR CODE HERE
    // Implement vector add. This kernel should add A and B and store the result in C.
}

int main()
{
    size_t size = N * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < N; i++)
    {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    hipError_t err = hipSuccess;
    float *d_A, *d_B, *d_C;
    err = hipMalloc(&d_A, size);
    err = hipMalloc(&d_B, size);
    err = hipMalloc(&d_C, size);

    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(THREADS_PER_BLOCK);
    dim3 numBlocks(N / threadsPerBlock.x);
    VecAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    bool error = false;
    for (int i = 0; i < N; i++)
    {
        // check if correct
        if (h_C[i] != h_A[i] + h_B[i])
        {
            error = true;
            printf("Error: %f + %f != %f\n", h_A[i], h_B[i], h_C[i]);
            break;
        }
    }
    if (!error)
    {
        printf("Success!\n");
    }
}
