#include <stdio.h>
#include <hip/hip_runtime.h>

#define N (1 << 20)

// Kernel definition
__global__ void VecAdd(float *A, float *B,
                       float *C)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    *(C + i) = *(A + i) + *(B + i);
}

int main()
{
    size_t size = N * sizeof(float);

    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    for (int i = 0; i < N; i++)
    {
        h_A[i] = i;
        h_B[i] = 2 * i;
    }

    hipError_t err = hipSuccess;
    float *d_A, *d_B, *d_C;
    err = hipMalloc(&d_A, size);
    err = hipMalloc(&d_B, size);
    err = hipMalloc(&d_C, size);

    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32);
    dim3 numBlocks(N / threadsPerBlock.x);
    printf("numBlocks: %d\n", numBlocks.x);
    VecAdd<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C);

    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    bool error = false;
    for (int i = 0; i < N; i++)
    {
        // check if correct
        if (h_C[i] != h_A[i] + h_B[i])
        {
            error = true;
            printf("Error: %f + %f != %f\n", h_A[i], h_B[i], h_C[i]);
            break;
        }
    }
    if (!error)
    {
        printf("Success!\n");
    }
}
